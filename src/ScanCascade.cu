#include "hip/hip_runtime.h"
﻿#include "ScanCascade.cuh"

using std::vector;
using thrust::device_vector;

ScanCascade::ScanCascade(
	int n_scan_units, int n_estimators, int n_ferns, int depth, int win_size, int stride)
	: scan_units(n_scan_units, ScanUnit(n_estimators, n_ferns, depth, win_size, stride))
{}

ScanCascade::ScanCascade(const json11::Json::array & config_array)
{
	scan_units.reserve(config_array.size());

	for (auto& config : config_array)
		scan_units.push_back(ScanUnit(
			config["N Random Ferns"].int_value(),
			config["N Ferns"].int_value(),
			config["depth"].int_value(),
			config["windows size"].int_value(),
			config["stride"].int_value()));
}


void ScanCascade::fit(const vector<vector<uint8_t>>& data, const vector<uint32_t>& labels, uint32_t batch_size)
{
	for (auto& unit : scan_units) unit.startFitting();

	device_vector<uint8_t> data_batch;
	device_vector<uint32_t> label_batch;
	uint32_t current_size;
	for (int i = 0; i < data.size(); i += batch_size) {
		current_size = std::min(static_cast<int>(batch_size), static_cast<int>(data.size()) - i);
		data_batch = packBatch(data, current_size, i);
		label_batch = packBatch(labels, current_size, i);

		for (auto& unit : scan_units) 
			unit.processBatch(data_batch, label_batch);

		hipDeviceSynchronize();
		data_batch = device_vector<uint8_t>();
		label_batch = device_vector<uint32_t>();
	}

	for (auto& unit : scan_units) unit.endFitting();
}

void ScanCascade::calculateTransform(const vector<vector<uint8_t>>& data, uint32_t batch_size)
{
	for (auto& unit : scan_units)
		unit.calculateTransform(data, batch_size);
}

void ScanCascade::clearTransformed()
{
	for (auto& unit : scan_units)
		unit.clearTransformed();
}

const vector<vector<float>>& ScanCascade::getTransformed(uint32_t index) const
{
	return scan_units[index].getTranformed();
}

void ScanCascade::setClassesNumber(uint32_t n_classes)
{
	for (auto& unit : scan_units) unit.setClassesNumber(n_classes);
}

void ScanCascade::setFeaturesNumber(uint32_t n_features)
{
	this->n_features = n_features;
	for (auto& unit : scan_units) unit.setFeaturesNumber(n_features);
}


device_vector<uint32_t> ScanCascade::packBatch(
	const vector<uint32_t>& in,
	uint32_t batch_size,
	uint32_t start_idx)
{
	device_vector<uint32_t> out(batch_size);
	thrust::copy(in.begin() + start_idx, in.begin() + start_idx + batch_size, out.begin());
	return out;
}

device_vector<uint8_t> ScanCascade::packBatch(
	const vector<vector<uint8_t>>& in,
	uint32_t batch_size,
	uint32_t start_idx)
{
	device_vector<uint8_t> out(batch_size * n_features);
	auto out_it = out.begin();
	auto data_it = in.begin() + start_idx;
	uint32_t sample_size = data_it->size();
	while (out_it != out.end())
	{
		thrust::copy(data_it->begin(), data_it->end(), out_it);
		out_it += sample_size;
		data_it++;
	}

	return out;
}